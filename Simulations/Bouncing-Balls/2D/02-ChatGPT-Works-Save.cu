#include "hip/hip_runtime.h"
// Written by ChatGPT Dec 15 Version
// I changed some variable names and constants
// for example they named yBot as floor which caused errors
// they also named yTop as ceiling instead
// Compiling worked for me: nvcc 02-ChatGPT.cu -o 2.out `pkg-config opencv4 --cflags --libs` && ./2.out
#include <iostream>
#include <cmath>
#include <array>
#include <random>
#include <opencv2/opencv.hpp>

constexpr int numBalls = 50;
constexpr double g = -9.81; // acceleration due to gravity
constexpr double dt = 0.2; // time step
constexpr int numTimeSteps = 30;
constexpr double yBot = 0; // y-coordinate of the yBot
constexpr double yTop = 360; // y-coordinate of the yTop
constexpr double xLeft = 0; // x-coordinate of the left wall
constexpr double xRight = 480; // x-coordinate of the right wall

struct Ball
{
    double x = 0; // x-coordinate of the center
    double y = 0; // y-coordinate of the center
    double vx = 0; // velocity in the x direction
    double vy = 0; // velocity in the y direction
    double m = 1; // mass
    double r = 8; // radius
};

std::array<Ball, numBalls> hostBalls; // balls on the host (CPU)
Ball* deviceBalls; // balls on the device (GPU)
std::mt19937 rng(std::random_device{}()); // random number generator
std::uniform_real_distribution<double> xDist(0.0, 480.0); // random x-coordinate generator
std::uniform_real_distribution<double> yDist(0.0, 360.0); // random y-coordinate generator
std::uniform_real_distribution<double> xVelo(-20.0, 20.0); // random x-coordinate generator
std::uniform_real_distribution<double> yVelo(-20.0, 20.0); // random y-coordinate generator

__global__ void updatePositions(Ball* balls, int numBalls, double dt, double g)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numBalls) return;

    Ball& ball = balls[i];
    ball.x += ball.vx * dt;
    ball.y += ball.vy * dt;
    ball.vy += -g * dt;
}

__global__ void bounceOffWalls(Ball* balls, int numBalls, double yBot, double yTop, double xLeft, double xRight)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numBalls) return;

    Ball& ball = balls[i];
    if (ball.y < yBot + ball.r)
    {
        ball.y = yBot + ball.r;
        ball.vy = std::abs(ball.vy);
    }
    else if (ball.y > yTop - ball.r)
    {
        ball.y = yTop - ball.r;
        ball.vy = -std::abs(ball.vy);
    }
    if (ball.x < xLeft + ball.r)
    {
        ball.x = xLeft + ball.r;
        ball.vx = std::abs(ball.vx);
    }
    else if (ball.x > xRight - ball.r)
    {
        ball.x = xRight - ball.r;
    ball.vx = -std::abs(ball.vx);
    }
}

int main()
{
    // allocate memory on the device
    hipMalloc((void**)&deviceBalls, numBalls * sizeof(Ball));

    // initialize the balls with random positions on the device
    for (int i = 0; i < numBalls; i++)
    {
        Ball ball;
        ball.x = xDist(rng);
        ball.y = yDist(rng);
        ball.vx = xVelo(rng);
        ball.vy = yVelo(rng);
        hipMemcpy(deviceBalls + i, &ball, sizeof(Ball), hipMemcpyHostToDevice);
    }

    // update positions and velocities
    for (int t = 0; t < numTimeSteps; t++)
    {
        // launch the updatePositions kernel
        int blockSize = 32;
        int numBlocks = (numBalls + blockSize - 1) / blockSize;
        updatePositions<<<numBlocks, blockSize>>>(deviceBalls, numBalls, dt, g);

        // launch the bounceOffWalls kernel
        bounceOffWalls<<<numBlocks, blockSize>>>(deviceBalls, numBalls, yBot, yTop, xLeft, xRight);

        // copy the device balls back to the host
        hipMemcpy(hostBalls.data(), deviceBalls, numBalls * sizeof(Ball), hipMemcpyDeviceToHost);

        // create an image with a white background
        cv::Mat image = cv::Mat::ones(360, 480, CV_8UC3) * 255;

        // draw the balls on the image
        for (int i = 0; i < numBalls; i++)
        {
            const Ball& ball = hostBalls[i];
            cv::circle(image, cv::Point(ball.x, ball.y), ball.r, cv::Scalar(0, 0, 0), -1);
        }

        // save the image
        cv::imwrite(std::to_string(t) + ".jpg", image);
    }

    // free memory on the device
    hipFree(deviceBalls);

    return 0;
}
