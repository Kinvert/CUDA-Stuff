// Written by ChatGPT Dec 15 Version#include <cuda_runtime.h>

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cstdlib>

// Activation function
__global__ void sigmoid(int n, double* x) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    x[i] = 1.0 / (1.0 + exp(-x[i]));
  }
}

// Matrix-vector product kernel
__global__ void matrix_vector_product(int m, int n, double* z, const double* bias, const double* weight, const double* input) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < m) {
    z[i] = bias[i];
    for (int j = 0; j < n; j++) {
      z[i] += weight[i * n + j] * input[j];
    }
  }
}

// Neural network class
class NeuralNetwork {
 public:
  // Constructor
  NeuralNetwork(const std::vector<int>& layer_sizes) : num_layers(layer_sizes.size()), biases(num_layers - 1), weights(num_layers - 1) {
    // Initialize biases and weights randomly
    for (int i = 0; i < num_layers - 1; i++) {
      int m = layer_sizes[i + 1];
      int n = layer_sizes[i];
      biases[i] = std::vector<double>(m);
      weights[i] = std::vector<std::vector<double>>(m, std::vector<double>(n));
      for (int j = 0; j < m; j++) {
        biases[i][j] = static_cast<double>(rand()) / RAND_MAX;
        for (int k = 0; k < n; k++) {
          weights[i][j][k] = static_cast<double>(rand()) / RAND_MAX;
        }
      }
    }
  }

  // Feedforward function
  std::vector<double> feedforward(const std::vector<double>& input) {
    int n = input.size();
    double* d_input;
    hipMalloc(&d_input, n * sizeof(double));
    hipMemcpy(d_input, input.data(), n * sizeof(double), hipMemcpyHostToDevice);
    std::vector<double> activations = input;
    // Propagate input through the layers
    for (int i = 0; i < num_layers - 1; i++) {
      int m = biases[i].size();
      
      // Note: ChatGPT tried to fix some of the issues with this
      // Note: But it just created a new segfault
      double* d_z;
      hipMalloc(&d_z, m * sizeof(double));
      hipMemset(d_z, 0, m * sizeof(double));

//      double* d_z = new double[m];
      double* d_bias = new double[m];
      double* d_weight = new double[m * n];
      double* d_output = new double[m];
      hipMemcpy(d_bias, biases[i].data(), m * sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d_weight, weights[i].data()->data(), m * n * sizeof(double), hipMemcpyHostToDevice);
      // Compute z = bias + input * weight
      matrix_vector_product<<<(m + 31) / 32, 32>>>(m, n, d_z, d_bias, d_weight, d_input);
      // Compute output = sigmoid(z)
      sigmoid<<<(m + 31) / 32, 32>>>(m, d_z);
      hipMemcpy(d_output, d_z, m * sizeof(double), hipMemcpyDeviceToHost);
      activations = std::vector<double>(d_output, d_output + m);
      delete[] d_z;
      delete[] d_bias;
      delete[] d_weight;
      delete[] d_output;
      hipFree(d_input);
      d_input = d_z;
    }
    hipFree(d_input);
    return activations;
  }

 private:
  int num_layers;
  std::vector<std::vector<double>> biases;
  std::vector<std::vector<std::vector<double>>> weights;
};

int main() {
  // Create neural network
  std::vector<int> layer_sizes = {2, 3, 2};
  NeuralNetwork nn(layer_sizes);

  // Input sample
  std::vector<double> input = {1.0, 2.0};

  // Compute output
  std::vector<double> output = nn.feedforward(input);

  // Print output
  for (double o : output) {
    std::cout << o << std::endl;
  }

  return 0;
}
