// Written by ChatGPT Dec 15 Version

#include <iostream>
#include <random>
#include <algorithm>
#include <cmath>

// Include the SGD kernel
#include "50-Kernel-ChatGPT-SGD.cu"

// Define the Example structure
struct Example {
  float x;
  float y;
};

const int NUM_EXAMPLES = 1000;
const int NUM_EPOCHS = 100;
const float LEARNING_RATE = 0.001;

// Generates a synthetic dataset with random noise added to the output
std::vector<Example> generate_synthetic_data(int num_examples) {
  std::vector<Example> examples;
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<float> dis(-10.0, 10.0);

  for (int i = 0; i < num_examples; i++) {
    float x = dis(gen);
    float y = 3.0 * x + 2.0 + dis(gen) * 0.1; // y = 3x + 2 + noise
    examples.push_back({x, y});
  }
  return examples;
}

int main() {
  std::vector<Example> examples = generate_synthetic_data(NUM_EXAMPLES);
  float weight = 0.0, bias = 0.0;

  // Allocate device memory for the weight and bias
  float* d_weight;
  hipMalloc((void**)&d_weight, sizeof(float));
  float* d_bias;
  hipMalloc((void**)&d_bias, sizeof(float));

  // Allocate device memory for the examples
  Example* d_examples;
  hipMalloc((void**)&d_examples, sizeof(Example) * NUM_EXAMPLES);

  // Copy the examples to the device
  hipMemcpy(d_examples, examples.data(), sizeof(Example) * NUM_EXAMPLES, hipMemcpyHostToDevice);

  for (int epoch = 0; epoch < NUM_EPOCHS; epoch++) {
    std::shuffle(examples.begin(), examples.end(), std::mt19937(std::random_device()()));

    // Set the number of threads and blocks for the kernel
    int block_size = 256;
    int num_blocks = (num_examples  + block_size - 1) / block_size;

  // Launch the kernel
  sgd_update_kernel<<<num_blocks, block_size>>>(d_examples, NUM_EXAMPLES, d_weight, d_bias, LEARNING_RATE);

  // Check for kernel launch errors
  hipError_t launch_status = hipGetLastError();
  if (launch_status != hipSuccess) {
    std::cerr << "Error launching kernel: " << hipGetErrorString(launch_status) << std::endl;
    return 1;
  }
}

// Copy the weight and bias back to the host
float weight, bias;
hipError_t weight_copy_status = hipMemcpy(&weight, d_weight, sizeof(float), hipMemcpyDeviceToHost);
hipError_t bias_copy_status = hipMemcpy(&bias, d_bias, sizeof(float), hipMemcpyDeviceToHost);

// Check for errors in the memory copies
if (weight_copy_status != hipSuccess || bias_copy_status != hipSuccess) {
  std::cerr << "Error copying weight or bias from device: " << hipGetErrorString(hipGetLastError()) << std::endl;
  return 1;
}

// Free the device memory
hipError_t weight_free_status = hipFree(d_weight);
hipError_t bias_free_status = hipFree(d_bias);
hipError_t examples_free_status = hipFree(d_examples);

// Check for errors in the memory freeing
if (weight_free_status != hipSuccess || bias_free_status != hipSuccess || examples_free_status != hipSuccess) {
  std::cerr << "Error freeing device memory: " << hipGetErrorString(hipGetLastError()) << std::endl;
  return 1;
}

std::cout << "Final weight: " << weight << std::endl;
std::cout << "Final bias: " << bias << std::endl;

  return 0;
}

