// ChatGPT wrote this
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// define the number of points to generate
const int num_points = 10000000;

// define the kernel function to estimate Pi
__global__ void estimate_pi(int* num_in_circle, hiprandState* states) {
  // get the index of the current thread
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  
  // initialize the cuRAND states
  hiprand_init(0, 0, 0, states);
  
  // generate a random point using the cuRAND library
  double x = hiprand_uniform(&states[i]);
  double y = hiprand_uniform(&states[i]);

  // check if the point lies within the unit circle
  if (x * x + y * y < 1.0) {
    // increment the number of points within the circle
    atomicAdd(num_in_circle, 1);
  }
}

int main() {
  // allocate memory on the host for the number of points in the circle
  int* num_in_circle_h;
  hipHostMalloc(&num_in_circle_h, sizeof(int), hipHostMallocDefault);

  // allocate memory on the device for the number of points in the circle
  int* num_in_circle_d;
  hipMalloc(&num_in_circle_d, sizeof(int));

  // initialize the number of points in the circle to 0
  *num_in_circle_h = 0;
  hipMemcpy(num_in_circle_d, num_in_circle_h, sizeof(int), hipMemcpyHostToDevice);

  // allocate memory on the device for the cuRAND states
  hiprandState* states_d;
  hipMalloc(&states_d, num_points * sizeof(hiprandState));

  // launch the kernel to estimate Pi
  estimate_pi<<<(num_points + 255) / 256, 256>>>(num_in_circle_d, states_d);

  // copy the result from the device to the host
  hipMemcpy(num_in_circle_h, num_in_circle_d, sizeof(int), hipMemcpyDeviceToHost);

  // compute and print the estimated value of Pi
  double pi = 4.0 * *num_in_circle_h / num_points;
  std::cout << "Estimated value of Pi: " << pi << std::endl;

  // free the memory allocated on the host and device
  hipFree(num_in_circle_d);
  hipFree(states_d);

  // free the memory on the host
  delete[] num_in_circle_h;

  return 0;
}
