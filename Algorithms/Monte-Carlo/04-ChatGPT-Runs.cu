// ChatGPT Dec 15 Version wrote this
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

__global__ void estimate_pi(int n, hiprandState* states, float* pi) {
  // get the index of the current thread
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // initialize the cuRAND states
  hiprand_init(0, i, 0, states);

  // generate a random point in the unit square
  float x = hiprand_uniform(&states[i]);
  float y = hiprand_uniform(&states[i]);

  // count how many points fall within the unit circle
  if (x * x + y * y < 1) {
    atomicAdd(pi, 1);
  }
}

int main() {
  // number of points to generate
  const int n = 100000000;

  // allocate device memory
  hiprandState* states;
  hipMalloc((void**)&states, n * sizeof(hiprandState));
  float* pi;
  hipMalloc((void**)&pi, sizeof(float));

  // create curand states on the device
  int block_size = 256;
  int num_blocks = (n + block_size - 1) / block_size;
//  curandCreateStates(states, n, 0);

  // launch kernel to generate random points and estimate pi
  estimate_pi<<<num_blocks, block_size>>>(n, states, pi);

  // copy result from device to host
  float pi_h;
  hipMemcpy(&pi_h, pi, sizeof(float), hipMemcpyDeviceToHost);

  // print result
  std::cout << "Estimated value of pi: " << 4.0 * pi_h / n << std::endl;

  // free device memory
  hipFree(states);
  hipFree(pi);

  return 0;
}
