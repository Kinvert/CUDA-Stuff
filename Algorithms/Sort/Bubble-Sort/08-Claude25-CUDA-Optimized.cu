// Written by Claude 2.5

#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>

#define N 65536

__global__ void bubbleSortShared(int *arr, int n) {

  __shared__ int sArr[512];

  int threadID = threadIdx.x;
  int blockID = blockIdx.x;

  int i = blockID * blockDim.x + threadID;

  sArr[threadID] = arr[i];
  __syncthreads();

  for(int pass=0; pass<n-1; pass++) {

    if(threadID < n-pass-1) {
      if(sArr[threadID] > sArr[threadID+1]) {
        int temp = sArr[threadID];
        sArr[threadID] = sArr[threadID+1];
        sArr[threadID+1] = temp;
      }
    }

    __syncthreads();
  }

  arr[i] = sArr[threadID];

}

int main() {

  int n = N;
  int *arr = new int[n];
  int *d_arr;

  // Initialize arr with random numbers
  srand(time(NULL));
  for (int i = 0; i < n; i++) {
    arr[i] = rand(); 
  }

  hipMalloc(&d_arr, n * sizeof(int));
  hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  bubbleSortShared<<<(n+255)/256, 256>>>(d_arr, n);
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);

  // Print sorted array
  hipMemcpy(arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);

  std::cout << "Sorted array: ";
  for (int i = 0; i < n; i++) {
    std::cout << arr[i] << " ";
  }

  std::cout << "\nTime taken: " << elapsedTime << " ms" << std::endl;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(d_arr);
  delete[] arr;

  return 0;
}
