// Written by ChatGPT
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <cstdlib>
#include <ctime>

// Function to perform insertion sort (C++ version)
void insertionSortCpp(int arr[], int n)
{
    for (int i = 1; i < n; i++)
    {
        int key = arr[i];
        int j = i - 1;
        while (j >= 0 && arr[j] > key)
        {
            arr[j + 1] = arr[j];
            j--;
        }
        arr[j + 1] = key;
    }
}

// Define the kernel function for insertion sort (CUDA version)
__global__ void insertionSortKernel(int* arr, int n)
{
    // Each thread sorts one element
    int i = threadIdx.x;

    // Insertion sort
    int key = arr[i];
    int j = i - 1;
    while (j >= 0 && arr[j] > key)
    {
        arr[j + 1] = arr[j];
        j--;
    }
    arr[j + 1] = key;
}

// Function to print the array
void printArray(int arr[], int n)
{
    for (int i = 0; i < n; i++)
        std::cout << arr[i] << " ";
    std::cout << std::endl;
}

int main()
{
    // Generate a random array of size 1000
    const int n = 1000;
    int arr[n];
    srand(time(0));
    for (int i = 0; i < n; i++)
        arr[i] = rand();

    // Make a copy of the array for the CUDA version
    int arr_cuda[n];
    for (int i = 0; i < n; i++)
        arr_cuda[i] = arr[i];

    // Sort the array using C++ insertion sort
    auto start_cpp = std::chrono::high_resolution_clock::now();
    insertionSortCpp(arr, n);
    auto end_cpp = std::chrono::high_resolution_clock::now();
    auto elapsedTimeCpp = std::chrono::duration_cast<std::chrono::milliseconds>(end_cpp - start_cpp).count();

    // Allocate memory on the device for the array for the CUDA version
    int* dev_arr;
    hipMalloc(&dev_arr, n * sizeof(int));

    // Copy the array from host to device for the CUDA version
    hipMemcpy(dev_arr, arr_cuda, n * sizeof(int), hipMemcpyHostToDevice);

    // Set the number of threads and blocks for the kernel
    const int num_threads = 256;
    const int num_blocks = (n + num_threads - 1) / num_threads;

    // Sort the array using CUDA insertion sort
    hipEvent_t start_cuda, end_cuda;
    hipEventCreate(&start_cuda);
    hipEventCreate(&end_cuda);
    hipEventRecord(start_cuda, 0);
    insertionSortKernel<<<num_blocks, num_threads>>>(dev_arr, n);
    hipEventRecord(end_cuda, 0);
    hipEventSynchronize(end_cuda);

    // Copy the sorted array from device to host for the CUDA version
    hipMemcpy(arr_cuda, dev_arr, n * sizeof(int), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(dev_arr);

    // Calculate elapsed time for CUDA version
    float elapsedTimeCuda;
    hipEventElapsedTime(&elapsedTimeCuda, start_cuda, end_cuda);

    // Print elapsed time for C++ and CUDA versions
    std::cout << "Elapsed time (C++): " << elapsedTimeCpp << " milliseconds" << std::endl;
    std::cout << "Elapsed time (CUDA): " << elapsedTimeCuda << " milliseconds" << std::endl;

    return 0;
}
