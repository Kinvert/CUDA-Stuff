#include "hip/hip_runtime.h"
// Written by ChatGPT Dec 15 Version

#include <iostream>
#include <cmath>
#include <opencv2/opencv.hpp>

using namespace std;
using namespace cv;

const int N = 100; // number of points
const int size = 280; // size of the image

struct Point {
    double x, y;
};

// function to check if a point lies inside a circle
__device__ bool isInside(Point p, Point c, double r)
{
    double dx = p.x - c.x;
    double dy = p.y - c.y;
    double d = sqrt(dx * dx + dy * dy);
    return d <= r;
}

int main()
{
    Point points[N]; // points on the plane

    // generate random points
    for (int i = 0; i < N; i++) {
        points[i].x = rand() % size;
        points[i].y = rand() % size;
    }

    double r = -1; // radius of the largest empty circle
    Point c; // center of the largest empty circle

    // allocate memory on the GPU
    Point *dev_points;
    hipMalloc((void **)&dev_points, N * sizeof(Point));
    hipMemcpy(dev_points, points, N * sizeof(Point), hipMemcpyHostToDevice);

    // launch a kernel to find the largest empty circle
    dim3 blocks(N, N);
    dim3 threads(1, 1);
    hipLaunchKernel(reinterpret_cast<const void*>(findLargestEmptyCircle), blocks, threads, 0, 0, dev_points, r, c);

    // copy the result back to the host
    hipMemcpy(&r, &r, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&c, &c, sizeof(Point), hipMemcpyDeviceToHost);

    // create an image and draw the points and circle
    Mat image(size, size, CV_8UC3, Scalar(0, 0, 0)); // create a black image
    for (int i = 0; i < N; i++) {
        circle(image, Point(points[i].x, points[i].y), 3, Scalar(0, 0, 255), -1); // draw a red point
    }
    circle(image, Point(c.x, c.y), r, Scalar(0, 255, 0), 2); // draw a green circle

    // show the image
    namedWindow("Largest empty circle", WINDOW_NORMAL);
    imshow("Largest empty circle", image);

    waitKey(0); // wait for a key press to exit

    return 0;
}

__global__ void findLargestEmptyCircle(Point *points, double r, Point c)
{
    int i = blockIdx.x;
    int j = blockIdx.y;
    if (i == j) return; // skip same points

    double dx = points[i].x - points[j].x;
    double dy = points[i].y - points[j].y;
    double rr = sqrt(dx * dx + dy * dy) / 2; // try radius as half the distance between i and j
    Point cc = {(points[i].x + points[j].x) / 2, (points[i].y + points[j].y) / 2}; // try center as midpoint of i and j

    // check if all other points are outside the circle
    bool valid = true;
    for (int k = 0; k < N; k++) {
        if (k == i || k == j) continue; // skip points i and j
        if (isInside(points[k], cc, rr)) { // check if point k is inside the circle
            valid = false;
            break;
        }
    }

    // update radius and center if valid and larger than current maximum
    if (valid && rr > r) {
        r = rr;
        c = cc;
    }
}

