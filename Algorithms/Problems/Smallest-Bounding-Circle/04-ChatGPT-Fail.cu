#include "hip/hip_runtime.h"
// Written by ChatGPT Jan 9 Version

#include <opencv2/opencv.hpp>
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime.h>

using namespace std;
using namespace cv;

__global__ void smallest_bounding_circle(Point2f *points, int n, Point2f *center, float *radius) {
    Point2f sum = make_float2(0, 0);
    for (int i = 0; i < n; i++) {
        sum.x += points[i].x;
        sum.y += points[i].y;
    }
    sum.x /= n;
    sum.y /= n;
    center[0] = sum;
    *radius = 0;
    for (int i = 0; i < n; i++) {
        float dist = sqrtf((points[i].x - sum.x) * (points[i].x - sum.x) + (points[i].y - sum.y) * (points[i].y - sum.y));
        *radius = max(*radius, dist);
    }
}

int main() {
    Mat img(280, 280, CV_8UC3, Scalar(0, 0, 0));
    vector<Point2f> points;
    RNG rng(12345);
    for (int i = 0; i < 50; i++) {
        Point2f point(rng.uniform(40, 240), rng.uniform(40, 240));
        points.push_back(point);
        circle(img, point, 2, Scalar(255, 0, 0), -1);
    }

    Point2f *d_points;
    Point2f *d_center;
    float *d_radius;
    hipMalloc((void **)&d_points, points.size() * sizeof(Point2f));
    hipMalloc((void **)&d_center, sizeof(Point2f));
    hipMalloc((void **)&d_radius, sizeof(float));
    hipMemcpy(d_points, &points[0], points.size() * sizeof(Point2f), hipMemcpyHostToDevice);

    smallest_bounding_circle<<<1, 1>>>(d_points, points.size(), d_center, d_radius);

    Point2f center;
    float radius;
    hipMemcpy(&center, d_center, sizeof(Point2f), hipMemcpyDeviceToHost);
    hipMemcpy(&radius, d_radius, sizeof(float), hipMemcpyDeviceToHost);

    circle(img, center, radius, Scalar(0, 255, 0), 2);
    imshow("Bounding Circle", img);
    waitKey(0);

    hipFree(d_points);
    hipFree(d_center);
    hipFree(d_radius);
    return 0;
}

