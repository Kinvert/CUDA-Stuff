#include "hip/hip_runtime.h"
// Works for me: nvcc 09-Race-Condition.cu -o 9.out `pkg-config opencv4 --cflags --libs` && ./4.out
// Race Condition
#include <cmath>
#include <ctime>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <random>
#include <stdio.h>

const int N = 100;
const int IMGSIZE = 280;

struct Point {
    double x = 0;
    double y = 0;
};

std::array<Point, N> h_p;
Point* d_p;
std::mt19937 rng(std::random_device{}());
std::uniform_real_distribution<double> dist(0.0, 280.0);

__device__ double distance(Point a, Point b) {
    return sqrt((a.x - b.x) * (a.x - b.x) + (a.y - b.y) * (a.y - b.y));
}

__global__ void closestPairPoints(int* a, int* b, Point* const p, double* res) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x != y && x < N && y < N) {
        double thisDist = distance(p[x], p[y]);
        if (thisDist < *res) {
            *res = thisDist;
            *a = x;
            *b = y;
        }
    }
    __syncthreads();
}

int main() {
    srand(time(0));
    
    hipMalloc((void**)&d_p, N * sizeof(Point));
    
    int* h_a = new int[1];
    int* h_b = new int[1];
    int* d_a;
    int* d_b;
    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));
    double* h_res = new double[1];
    double* d_res;
    hipMalloc(&d_res, sizeof(double));
    
    *h_res = 65353.0;
    *h_a = 999;
    *h_b = 999;
    
    for (int i = 0; i < N; i++) {
        Point& p = h_p[i];
        p.x = dist(rng);
        p.y = dist(rng);
    }
    
    hipMemcpy(d_p, h_p.data(), N * sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_a, h_a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_res, h_res, sizeof(double), hipMemcpyHostToDevice);
    
    dim3 gridDim(2, 2, 1);
    dim3 blockDim(32, 32, 1);
    
    printf("BEFORE KERNEL\n");
    closestPairPoints<<<gridDim, blockDim>>>(d_a, d_b, d_p, d_res);
    hipDeviceSynchronize();
    printf("Before Device to Host\n");
    hipMemcpy(h_a, d_a, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_b, d_b, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_res, d_res, sizeof(double), hipMemcpyDeviceToHost);
    
    cv::Mat image(IMGSIZE, IMGSIZE, CV_8UC3, cv::Scalar(0, 0, 0));
    for (int i = 0; i < N; i++) {
        cv::Scalar color = cv::Scalar(0, 0, 0);
        if (i == *h_a || i == *h_b) {
            color = cv::Scalar(0, 255, 0);
        } else {
            color = cv::Scalar(255, 0, 0);
        }
        cv::circle(image, cv::Point(int(h_p[i].x), int(h_p[i].y)), 2, color, -1); // draw a red point
    }
    
    printf("Result = %.3f\n", *h_res);
    cv::namedWindow("Largest empty circle", cv::WINDOW_NORMAL);
    cv::imshow("Largest empty circle", image);
    cv::imwrite("09-Result.png", image);
    cv::waitKey(0); // wait for a key press to exit
    
    return 0;
}

