#include "hip/hip_runtime.h"
// Written by me
#include <stdio.h>

// CUDA Search Kernel
__global__ void gpuLinearSearchKernel(int* arr, int numVals, int target, int* index)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numVals) {
        if (arr[i] == target) {
            printf("FOUND %d AT %d\n", target, i);
            *index = i;
        }
    }
    __syncthreads();
}

int main()
{
    // Array to GPU
    int numVals = 8;
    int h_arr[numVals] = { 4, 2, 6, 1, 3, 7, 8, 5 };
    int target = 5;
    int* g_arr;    
    hipMalloc(&g_arr, numVals * sizeof(int));
    hipMemcpy(g_arr, h_arr, numVals * sizeof(int), hipMemcpyHostToDevice);

    // Index to GPU
    int idx = -1;
    int* h_index;
    h_index = &idx;
    int* g_index;
    hipMalloc(&g_index, sizeof(int));
    hipMemcpy(g_index, h_index, sizeof(int), hipMemcpyHostToDevice);

    // Do the Search
    gpuLinearSearchKernel<<<1,numVals>>>(g_arr, numVals, target, g_index);

    // Index from GPU to CPU
    hipMemcpy(h_index, g_index, sizeof(int), hipMemcpyDeviceToHost);

    // Free Memory
    hipFree(g_arr);
    hipFree(g_index);

    // Print Results
    if (*h_index != -1) {
        printf("Target found at index: %d\n", *h_index);
    }
    else {
        printf("Target not found\n");
    }
    
    return 0;
}
