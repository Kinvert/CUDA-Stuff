// ChatGPT Dec 15 Version wrote this
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

// Kernel function to search for the target on the GPU
__global__
void gpuLinearSearchKernel(int* arr, int numVals, int target, int* index)
{
    // search for target in the array
    for (int i = 0; i < numVals; i++)
    {
        // if target is found, set the index and return
        if (arr[i] == target)
        {
            *index = i;
            return;
        }
    }
}

int main()
{
    std::vector<int> arr = { 4, 2, 6, 1, 3, 7, 8, 5 };
    int target = 5;

    // get the size of the vector
    int numVals = arr.size();

    // allocate memory on the GPU
    int* gpu_arr;
    hipMalloc((void**)&gpu_arr, numVals * sizeof(int));

    // copy data from the host to the GPU
    hipMemcpy(gpu_arr, arr.data(), numVals * sizeof(int), hipMemcpyHostToDevice);

    // launch the kernel to search for the target on the GPU
    int index = -1;
    gpuLinearSearchKernel<<<1,1>>>(gpu_arr, numVals, target, &index);

    // copy the result back to the host
    hipMemcpy(&index, gpu_arr, sizeof(int), hipMemcpyDeviceToHost);

    // free memory on the GPU
    hipFree(gpu_arr);

    if (index != -1)
        std::cout << "Target found at index: " << index << std::endl;
    else
        std::cout << "Target not found" << std::endl;

    return 0;
}
