#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1000  // Size of the data array

// Kernel function
__global__ void hello_kernel(int *data, int size)
{
    // Calculate the thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Only write to the data array if the thread index is within the bounds of the array
    if (i < size)
    {
        data[i] = i;
    }
}

int main()
{
    // Allocate device memory for the data array
    int *data;
    hipMalloc((void **)&data, N * sizeof(int));

    // Launch the kernel
    hello_kernel<<<1, N>>>(data, N);
    hipDeviceSynchronize();

    // Allocate host memory and copy data from the device
    int *host_data = (int *)malloc(N * sizeof(int));
    hipMemcpy(host_data, data, N * sizeof(int), hipMemcpyDeviceToHost);

    // Print the contents of the data array
    for (int i = 0; i < N; i++)
    {
        printf("%d ", host_data[i]);
    }
    printf("\n");

    // Free device and host memory
    hipFree(data);
    free(host_data);

    return 0;
}
